#include "hip/hip_runtime.h"
// Copyright 2024 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "lidar_transfusion/cuda_utils.hpp"
#include "lidar_transfusion/preprocess/preprocess_kernel.hpp"

namespace lidar_transfusion
{

PreprocessCuda::PreprocessCuda(const TransfusionConfig & config, hipStream_t & stream)
: stream_(stream), config_(config)
{
  mask_size_ = config_.grid_z_size_ * config_.grid_y_size_ * config_.grid_x_size_;
  voxels_size_ = config_.grid_z_size_ * config_.grid_y_size_ * config_.grid_x_size_ *
                   config_.max_num_points_per_pillar_ * config_.num_point_feature_size_ +
                 1;
  mask_ = cuda::make_unique<unsigned int[]>(mask_size_);
  voxels_ = cuda::make_unique<float[]>(voxels_size_);
}

void PreprocessCuda::generateVoxels(
  float * points, unsigned int points_size, unsigned int * pillar_num, float * voxel_features,
  unsigned int * voxel_num, unsigned int * voxel_idxs)
{
  cuda::clear_async(mask_.get(), mask_size_, stream_);
  cuda::clear_async(voxels_.get(), voxels_size_, stream_);
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream_));

  CHECK_CUDA_ERROR(generateVoxels_random_launch(points, points_size, mask_.get(), voxels_.get()));
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream_));

  CHECK_CUDA_ERROR(generateBaseFeatures_launch(
    mask_.get(), voxels_.get(), pillar_num, voxel_features, voxel_num, voxel_idxs));
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream_));
}

__global__ void generateVoxels_random_kernel(
  float * points, unsigned int points_size, float min_x_range, float max_x_range, float min_y_range,
  float max_y_range, float min_z_range, float max_z_range, float pillar_x_size, float pillar_y_size,
  float pillar_z_size, int grid_y_size, int grid_x_size, int points_per_voxel, unsigned int * mask,
  float * voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  float x = points[point_idx * 5];
  float y = points[point_idx * 5 + 1];
  float z = points[point_idx * 5 + 2];
  float i = points[point_idx * 5 + 3];
  float t = points[point_idx * 5 + 4];

  if (
    x <= min_x_range || x >= max_x_range || y <= min_y_range || y >= max_y_range ||
    z <= min_z_range || z >= max_z_range)
    return;

  int voxel_idx = floorf((x - min_x_range) / pillar_x_size);
  int voxel_idy = floorf((y - min_y_range) / pillar_y_size);
  unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if (point_id >= points_per_voxel) return;
  float * address = voxels + (voxel_index * points_per_voxel + point_id) * 5;
  atomicExch(address + 0, x);
  atomicExch(address + 1, y);
  atomicExch(address + 2, z);
  atomicExch(address + 3, i);
  atomicExch(address + 4, t);
}

hipError_t PreprocessCuda::generateVoxels_random_launch(
  float * points, unsigned int points_size, unsigned int * mask, float * voxels)
{
  int threadNum = config_.threads_for_voxel_;
  dim3 blocks((points_size + threadNum - 1) / threadNum);
  dim3 threads(threadNum);
  generateVoxels_random_kernel<<<blocks, threads, 0, stream_>>>(
    points, points_size, config_.min_x_range_, config_.max_x_range_, config_.min_y_range_,
    config_.max_y_range_, config_.min_z_range_, config_.max_z_range_, config_.voxel_x_size_,
    config_.voxel_y_size_, config_.voxel_z_size_, config_.grid_y_size_, config_.grid_x_size_,
    config_.points_per_voxel_, mask, voxels);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateBaseFeatures_kernel(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, float points_per_voxel,
  float max_voxels, unsigned int * pillar_num, float * voxel_features, unsigned int * voxel_num,
  unsigned int * voxel_idxs)
{
  unsigned int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int voxel_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if (voxel_idx >= grid_x_size || voxel_idy >= grid_y_size) return;

  unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;
  unsigned int count = mask[voxel_index];
  if (!(count > 0)) return;
  count = count < points_per_voxel ? count : points_per_voxel;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num, 1);
  if (current_pillarId >= max_voxels) return;

  voxel_num[current_pillarId] = count;

  uint4 idx = {0, 0, voxel_idy, voxel_idx};
  ((uint4 *)voxel_idxs)[current_pillarId] = idx;

  for (int i = 0; i < count; i++) {
    int inIndex = voxel_index * points_per_voxel + i;
    int outIndex = current_pillarId * points_per_voxel + i;
    voxel_features[outIndex * 5] = voxels[inIndex * 5];
    voxel_features[outIndex * 5 + 1] = voxels[inIndex * 5 + 1];
    voxel_features[outIndex * 5 + 2] = voxels[inIndex * 5 + 2];
    voxel_features[outIndex * 5 + 3] = voxels[inIndex * 5 + 3];
    voxel_features[outIndex * 5 + 4] = voxels[inIndex * 5 + 4];
  }

  // clear buffer for next infer
  atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t PreprocessCuda::generateBaseFeatures_launch(
  unsigned int * mask, float * voxels, unsigned int * pillar_num, float * voxel_features,
  unsigned int * voxel_num, unsigned int * voxel_idxs)
{
  dim3 threads = {32, 32};
  dim3 blocks = {divup(config_.grid_x_size_, threads.x), divup(config_.grid_y_size_, threads.y)};

  generateBaseFeatures_kernel<<<blocks, threads, 0, stream_>>>(
    mask, voxels, config_.grid_y_size_, config_.grid_x_size_, config_.points_per_voxel_,
    config_.max_voxels_, pillar_num, voxel_features, voxel_num, voxel_idxs);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateSweepPoints_kernel(
  const float * input_points, size_t points_size, int input_point_step, float time_lag,
  const float * transform_array, int num_features, float * output_points)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  const float input_x = input_points[point_idx * input_point_step + 0];
  const float input_y = input_points[point_idx * input_point_step + 1];
  const float input_z = input_points[point_idx * input_point_step + 2];
  const float intensity = input_points[point_idx * input_point_step + 3];

  output_points[point_idx * num_features] = transform_array[0] * input_x +
                                            transform_array[4] * input_y +
                                            transform_array[8] * input_z + transform_array[12];
  output_points[point_idx * num_features + 1] = transform_array[1] * input_x +
                                                transform_array[5] * input_y +
                                                transform_array[9] * input_z + transform_array[13];
  output_points[point_idx * num_features + 2] = transform_array[2] * input_x +
                                                transform_array[6] * input_y +
                                                transform_array[10] * input_z + transform_array[14];
  output_points[point_idx * num_features + 3] = intensity;
  output_points[point_idx * num_features + 4] = time_lag;
}

hipError_t PreprocessCuda::generateSweepPoints_launch(
  const float * input_points, size_t points_size, int input_point_step, float time_lag,
  const float * transform_array, float * output_points)
{
  int threadNum = config_.threads_for_voxel_;
  dim3 blocks((points_size + threadNum - 1) / threadNum);
  dim3 threads(threadNum);

  generateSweepPoints_kernel<<<blocks, threads, 0, stream_>>>(
    input_points, points_size, input_point_step, time_lag, transform_array,
    config_.num_point_feature_size_, output_points);

  hipError_t err = hipGetLastError();
  return err;
}

}  // namespace lidar_transfusion
